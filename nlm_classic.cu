#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKDIM_X 32
#define BLOCKDIM_Y 32

__device__ float Max(float x, float y)
{
    return (x > y) ? x : y;
}

__device__ float Min(float x, float y)
{
    return (x < y) ? x : y;
}

__device__ int Max(int x, int y)
{
    return (x > y) ? x : y;
}

__device__ int Min(int x, int y)
{
    return (x < y) ? x : y;
}

int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

//__global__ void nlm_classic_device2(const float* d_src, float* d_dst, float fSigma, float fParam, int patch, int window, int w, int h) {

//    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
//    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
//    if (ix < w && iy < h)
//    {
//        d_dst[w*iy+ix] = d_src[w*iy+ix];
//    }
//}

__global__ void nlm_classic_global(const float* d_src,
                                   float* d_dst,
                                   float fSigma,
                                   float fParam,
                                   int patch, int window,
                                   int w, int h,
                                   int patchSize, float fSigma2, float fH2, float icwl) {

    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    //Add half of a texel to always address exact texel centers
    //    const float x = (float)ix + 0.5f;
    //    const float y = (float)iy + 0.5f;

    if (ix < w && iy < h)
    {
        int i1 = ix+patch;
        int j1 = iy+patch;
        float wmax = 0;
        float average = 0;
        float sweight = 0;

        int rmin = Max(i1-window,patch);
        int rmax = Min(i1+window,w+patch);
        int smin = Max(j1-window,patch);
        int smax = Min(j1+window,h+patch);

        for (int r = rmin; r < rmax; r++) {
            for (int s = smin; s < smax; s++) {
                if (r == i1 && s == j1) {
                    continue;
                }
                float diff = 0;
                for (int ii = -patch; ii <= patch; ii++) {
                    for (int jj = -patch; jj <= patch; jj++) {
                        float a = d_src[w*(j1+jj)+(i1+ii)];
                        float b = d_src[w*(s+jj)+(r+ii)];
                        float c = a-b;
                        diff += c*c;
                    }
                }
                diff = Max(float(diff - 2.0 * (double) icwl *  fSigma2), 0.0f);
                diff = diff / fH2;
                float W = __expf(-diff);

                if (W > wmax) {
                    wmax = W;
                }

                sweight += W;
                average += W * d_src[w*s + r];
            }
        }
        average += wmax * d_src[w*j1+i1];
        sweight += wmax;

        if (sweight > 0) {
            d_dst[w*iy+ix] = average / sweight;
        }
        else {
            d_dst[w*iy+ix] = d_src[w*iy+ix];
        }
    }

}

void nlm_filter_classic_CUDA(const float* h_src, float* h_dst, int w, int h, float fSigma, float fParam, int patch, int window) {
    hipError_t err = hipSuccess;

    float* d_src = NULL, *d_dst = NULL;
    unsigned int nBytes = sizeof(float) * (w*h);

    err = hipMalloc((void **)& d_src, nBytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector SRC (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)& d_dst, nBytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector DST (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_src, h_src, nBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector SRC from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_dst, h_dst, nBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector DST from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(w, BLOCKDIM_X), iDivUp(h, BLOCKDIM_Y));

    int patchSize = patch*2+1;
    float fSigma2 = fSigma * fSigma;
    float fH = fParam * fSigma;
    float fH2 = fH * fH;
    float icwl = patchSize * patchSize;
    fH2 *= icwl;

    nlm_classic_global<<<grid, threads>>>(d_src, d_dst, fSigma, fParam, patch, window, w, h, patchSize, fSigma2, fH2, icwl);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch nlm_classic_device kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // load the answer back into the host
    err = hipMemcpy(h_dst, d_dst, nBytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector DST from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(d_src);
    hipFree(d_dst);
}
