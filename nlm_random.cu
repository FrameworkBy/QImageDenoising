#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#include <algorithm>
#include <vector>

#define BLOCKDIM_X 32
#define BLOCKDIM_Y 32

int iDivUp2(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

struct PatchDist {
    int iX;
    int iY;
    double fDist;
};

bool compareDist(PatchDist d1, PatchDist d2) {
    return d1.fDist < d2.fDist;
}

class PriorityQueue {
public:
    PriorityQueue() {
    }

    int size() {
        return pqDist.size();
    }

    void add(int x, int y, double dist) {
        PatchDist pd;
        pd.iX = x;
        pd.iY = y;
        pd.fDist = dist;
        if (pqDist.size() == 0 || pqDist.size() <= ss) {
            pqDist.push_back(pd);
            std::sort(pqDist.begin(), pqDist.end(), compareDist);
        } else {
            PatchDist pqLast = pqDist.at(pqDist.size()-1);
            if (pqDist.size() < ss || pqLast.fDist > dist) {
                pqDist.pop_back();
                pqDist.push_back(pd);
                std::sort(pqDist.begin(), pqDist.end(), compareDist);
            }
        }
    }

    PatchDist get(int pos) {
        return pqDist.at(pos);
    }

    void clear() {
        pqDist.clear();
    }

private:
    int ss = 11;
    std::vector<PatchDist> pqDist;
};


__device__ int2 randPoint(int cx, int cy, int sigma, int i) {
    hiprandState randState;
    unsigned int seed = (unsigned int) clock64();
    hiprand_init(seed , 0, 0, &randState);
    float2 fpoint;
    fpoint = hiprand_normal2(&randState);
    double k = 1;
    if (i != 0)
        k  = pow(0.5, i);
    int2 ipoint;
    ipoint.x = fpoint.x*100.0f;//cx + int(sigma * k * fpoint.x);
    ipoint.y = fpoint.y*100.0f;//cy + int(sigma * k * fpoint.y);
    return ipoint;
}

__global__ void getRand(int2 *rnd) {
    int2 r = randPoint(5, 3, 30, 0);
    rnd->x = r.x;
    rnd->y = r.y;
}

__global__ void initQueue(const float *d_src, int width, int height,
                          int patchRadius, int searchRadius, int queueSize) {

}

__global__ void nlm_filter_random_global(const float *d_src, float* d_dst, int width, int height,
                                         float fSigma, float fParam, int patchRadius,
                                         int searchRadius, int queueSize, int steps) {
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    if (ix < width && iy < height)
    {
        int i1 = ix+patchRadius;
        int j1 = iy+patchRadius;

        d_dst[width*j1 + i1] += d_src[width*j1 + i1]/(float)steps;
    }
}

void nlm_filter_random_CUDA(const float* h_src, float* h_dst,
                            int width, int height,
                            float fSigma, float fParam,
                            int patchRadius, int searchRadius,
                            int queueSize, int steps) {
    hipError_t err = hipSuccess;

    float* d_src = NULL, *d_dst = NULL;
    unsigned int nBytes = sizeof(float) * (width*height);

    err = hipMalloc((void **)& d_src, nBytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector SRC (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)& d_dst, nBytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector DST (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_src, h_src, nBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector SRC from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_dst, h_dst, nBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector DST from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp2(width, BLOCKDIM_X), iDivUp2(height, BLOCKDIM_Y));

    int2 *p, *h_p;
    h_p = (int2*) malloc(sizeof(int2));
    hipMalloc((void **)& p, sizeof(int2));

    for (int i = 0; i < steps; ++i) {
        getRand<<<1, 1>>>(p);

        hipMemcpy(h_p, p, sizeof(int2), hipMemcpyDeviceToHost);
        printf("%f\t%f\n", h_p->x, h_p->y);

        initQueue<<<grid, threads>>>(d_src, width, height, patchRadius, searchRadius, queueSize);
        nlm_filter_random_global<<<grid, threads>>>(d_src, d_dst, width, height,
                                                    fSigma, fParam, patchRadius, searchRadius, queueSize, steps);
    }

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch nlm_random_device kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // load the answer back into the host
    err = hipMemcpy(h_dst, d_dst, nBytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector DST from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(d_src);
    hipFree(d_dst);
}
