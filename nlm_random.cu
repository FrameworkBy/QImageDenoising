#include <hip/hip_runtime.h>
#include <stdio.h>

#include <algorithm>
#include <vector>

#define BLOCKDIM_X 8
#define BLOCKDIM_Y 8

struct PatchDist {
    int iX;
    int iY;
    double fDist;
};

bool compareDist(PatchDist d1, PatchDist d2) {
    return d1.fDist < d2.fDist;
}

class PriorityQueue {
public:
    PriorityQueue() {
    }

    int size() {
        return pqDist.size();
    }

    void add(int x, int y, double dist) {
        PatchDist pd;
        pd.iX = x;
        pd.iY = y;
        pd.fDist = dist;
        if (pqDist.size() == 0 || pqDist.size() <= ss) {
            pqDist.push_back(pd);
            std::sort(pqDist.begin(), pqDist.end(), compareDist);
        } else {
            PatchDist pqLast = pqDist.at(pqDist.size()-1);
            if (pqDist.size() < ss || pqLast.fDist > dist) {
                pqDist.pop_back();
                pqDist.push_back(pd);
                std::sort(pqDist.begin(), pqDist.end(), compareDist);
            }
        }
    }

    PatchDist get(int pos) {
        return pqDist.at(pos);
    }

    void clear() {
        pqDist.clear();
    }

private:
    int ss = 11;
    std::vector<PatchDist> pqDist;
};

void nlm_filter_random_CUDA(const float* h_src, float* h_dst, int w, int h, float fSigma, float fParam, int patch, int window) {
    hipError_t err = hipSuccess;
}
